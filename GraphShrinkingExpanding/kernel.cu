#include "hip/hip_runtime.h"
/*
NOTE: Run in VS using x64 platform.

TODO:

SHRINKING:
- Look into edge based vs CSR based device.
- Load graph should be a separate method

EXPANDING:
- Option for undirected (edge interconnection)
- Expanding fraction (e.g. 3.5) - what about the 0.5
- Investigate stream expanding
- Make it somewhat nice so that the user can change these properties easily.
- Decrease size of char in Bridge_Edge

ANALYSIS
- Check snap tool

OVERALL
- Refactor code (multiple files, remove code duplicates)
- Get rid of using mixed C/C++
*/

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <nvgraph.h>
#include "hip/device_functions.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <random>
#include <unordered_set>
#include <unordered_map>
#include <map>
#include <algorithm>

#define MAX_THREADS 1024
#define ENABLE_DEBUG_LOG false

int SIZE_VERTICES;
int SIZE_EDGES;
bool IS_INPUT_FILE_COO = false;

typedef enum Bridge_Node_Selection {HIGH_DEGREE_NODES, RANDOM_NODES} Bridge_Node_Selection;
typedef enum Topology {STAR, CHAIN, CIRCLE, MESH};
bool FORCE_UNDIRECTED_BRIDGES = false;
float SAMPLING_FRACTION;
float EXPANDING_FACTOR;
int AMOUNT_INTERCONNECTIONS;

Bridge_Node_Selection SELECTED_BRIDGE_NODE_SELECTION;
Topology SELECTED_TOPOLOGY;

typedef struct Sampled_Vertices sampled_vertices;
typedef struct COO_List coo_list;
typedef struct CSR_List csr_list;
typedef struct Edge edge;
typedef struct Sampled_Graph_Version;
typedef struct Bridge_Edge;
void load_graph_from_edge_list_file(int*, int*, char*);
COO_List* load_graph_from_edge_list_file_to_coo(std::vector<int>&, std::vector<int>&, char*);
int add_vertex_as_coordinate(std::vector<int>&, std::unordered_map<int, int>&, int, int);
int get_thread_size();
int calculate_node_sampled_size(float);
int get_block_size();
Sampled_Vertices* perform_edge_based_node_sampling_step(int*, int*, float);
void print_debug_log(char*);
void print_debug_log(char*, int);
void print_coo(int*, int*);
void print_csr(int*, int*);
void sample_graph(char*, char*, float);
CSR_List* convert_coo_to_csr_format(int*, int*);
void expand_graph(char*, char*, float);
void link_using_star_topology(Sampled_Graph_Version*, int, std::vector<Bridge_Edge>&);
void link_using_line_topology(Sampled_Graph_Version*, int, std::vector<Bridge_Edge>&);
void link_using_circle_topology(Sampled_Graph_Version*, int, std::vector<Bridge_Edge>&);
void link_using_mesh_topology(Sampled_Graph_Version*, int, std::vector<Bridge_Edge>&);
void add_edge_interconnection_between_graphs(Sampled_Graph_Version*, Sampled_Graph_Version*, std::vector<Bridge_Edge>&);
int select_random_bridge_vertex(Sampled_Graph_Version*);
int select_high_degree_node_bridge_vertex(Sampled_Graph_Version*);
int get_random_high_degree_node(Sampled_Graph_Version*);
void collect_sampling_parameters(char* argv[]);
void collect_expanding_parameters(char* argv[]);
void write_expanded_output_to_file(Sampled_Graph_Version*, int, std::vector<Bridge_Edge>&, char*);
void write_output_to_file(std::vector<Edge>&, char* output_path);
void save_input_file_as_coo(std::vector<int>&, std::vector<int>&, char*);
int get_node_bridge_vertex(Sampled_Graph_Version*);
void check(nvgraphStatus_t);

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

typedef struct COO_List {
	int* source;
	int* destination;
} COO_List;

typedef struct CSR_List {
	int* offsets;
	int* indices;
} CSR_List;

typedef struct Sampled_Vertices {
	int* vertices;
	int sampled_vertices_size;
} Sampled_Vertices;

typedef struct Edge {
	int source, destination;
} Edge;

typedef struct Sampled_Graph_Version {
	std::vector<Edge> edges;
	std::vector<int> high_degree_nodes;
	char label;
} Sampled_Graph_Version;

typedef struct Bridge_Edge {
	char source[20];
	char destination[20];
} Bridge_Edge;

__device__ int d_edge_count = 0;
__constant__ int D_SIZE_EDGES;
__constant__ int D_SIZE_VERTICES;

__device__ int push_edge(Edge &edge, Edge* d_edge_data) {
	int edge_index = atomicAdd(&d_edge_count, 1);
	if (edge_index < D_SIZE_EDGES) {
		d_edge_data[edge_index] = edge;
		return edge_index;
	}
	else {
		printf("Maximum edge size threshold reached: %d", D_SIZE_EDGES);
		return -1;
	}
}

__global__
void perform_induction_step(int* sampled_vertices, int* offsets, int* indices, Edge* d_edge_data) {
	int neighbor_index_start_offset = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (neighbor_index_start_offset < D_SIZE_VERTICES) {
		int neighbor_index_end_offset = neighbor_index_start_offset + 1;

		for (int n = offsets[neighbor_index_start_offset]; n < offsets[neighbor_index_end_offset]; n++) {
			if (sampled_vertices[neighbor_index_start_offset] && sampled_vertices[indices[n]]) {
				//printf("\nAdd edge: (%d,%d).", neighbor_index_start_offset, indices[n]);
				Edge edge;
				edge.source = neighbor_index_start_offset;
				edge.destination = indices[n];
				push_edge(edge, d_edge_data);
			}
		}
	}
}

clock_t t1;
clock_t t2;
clock_t total_t;

void perform_sequential_induction_step(int* sampled_vertices, int* offsets, int* indices, std::vector<Edge>& edges) {
	t1 = clock();
	for (int p = 0; p < SIZE_VERTICES; p++) {
		//printf("\n\nVertex %d", p);
		
		int startOffset = offsets[p];
		int endOffset = offsets[p + 1];
		//printf("\nHas neighbor:");
		for (int i = startOffset; i < endOffset ; i++) {
			//printf("%d, ", indices[i]);
			if (sampled_vertices[p] && sampled_vertices[indices[i]]) {
				//printf("\nAdd edge: (%d, %d)", p, indices[i]);
				Edge edge;
				edge.source = p;
				edge.destination = indices[i];
				edges.push_back(edge);
			}
		}
	}
	t2 = clock() - t1;
	printf("It took me %d clicks (%f seconds).\n", t2, ((float)t2) / CLOCKS_PER_SEC);
}

__device__ int push_edge_expanding(Edge &edge, Edge* edge_data_expanding, int* d_edge_count_expanding) {
	int edge_index = atomicAdd(d_edge_count_expanding, 1);
	if (edge_index < D_SIZE_EDGES) {
		edge_data_expanding[edge_index] = edge;
		return edge_index;
	}
	else {
		printf("Maximum edge size threshold reached.");
		return -1;
	}
}

__global__
void perform_induction_step_expanding(int* sampled_vertices, int* offsets, int* indices, Edge* edge_data_expanding, int* d_edge_count_expanding) {
	int neighbor_index_start_offset = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (neighbor_index_start_offset < D_SIZE_VERTICES) {
		int neighbor_index_end_offset = neighbor_index_start_offset + 1;

		for (int n = offsets[neighbor_index_start_offset]; n < offsets[neighbor_index_end_offset]; n++) {
			if (sampled_vertices[neighbor_index_start_offset] && sampled_vertices[indices[n]]) {
				//printf("\nAdd edge: (%d,%d).", neighbor_index_start_offset, indices[n]);
				Edge edge;
				edge.source = neighbor_index_start_offset;
				edge.destination = indices[n];
				push_edge_expanding(edge, edge_data_expanding, d_edge_count_expanding);
			}
		}
	}
}

int main(int argc, char* argv[]) {
	if (argc >= 4) {
		char* input_path = argv[1];
		char* output_path = argv[2];

		if (strcmp(argv[3], "sample") == 0) {
			collect_sampling_parameters(argv);
			sample_graph(input_path, output_path, SAMPLING_FRACTION);
		}
		else {
			collect_expanding_parameters(argv);
			expand_graph(input_path, output_path, EXPANDING_FACTOR);
		}
	} else {
		printf("Incorrect amount of input/output arguments given.");

		// ONLY FOR LOCAL TESTING
		//char* input_path = "C:\\Users\\AJ\\Documents\\example_graph.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\nvgraphtest\\nvGraphExample-master\\nvGraphExample\\web-Stanford.txt";
		char* input_path = "C:\\Users\\AJ\\Desktop\\nvgraphtest\\nvGraphExample-master\\nvGraphExample\\web-Stanford_large.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\edge_list_example.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\roadnet.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\facebook_graph.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\output_test\\social\\soc-pokec-relationships.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\roadNet-PA.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\soc-pokec-relationships.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\com-orkut.ungraph.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\soc-LiveJournal1.txt";
		//char* input_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\coo\\pokec_coo.txt";
		char* output_path = "C:\\Users\\AJ\\Desktop\\new_datasets\\output\\performance_testing.txt";

		/*sample_graph(input_path, output_path, 0.5);
		*/
		EXPANDING_FACTOR = 3;
		SAMPLING_FRACTION = 0.5;
		SELECTED_TOPOLOGY = STAR;
		SELECTED_BRIDGE_NODE_SELECTION = RANDOM_NODES;
		AMOUNT_INTERCONNECTIONS = 10;
		FORCE_UNDIRECTED_BRIDGES = true;
		expand_graph(input_path, output_path, EXPANDING_FACTOR);
	}

	return 0;
}

void collect_sampling_parameters(char* argv[]) {
	float fraction = atof(argv[4]);
	SAMPLING_FRACTION = fraction;
	printf("\nSample fraction: %f", fraction);
}

void collect_expanding_parameters(char* argv[]) {
	// Factor
	EXPANDING_FACTOR = atof(argv[4]);
	printf("\nFactor: %f", EXPANDING_FACTOR);

	// Fraction
	SAMPLING_FRACTION = atof(argv[5]);
	printf("\nFraction per sample: %f", SAMPLING_FRACTION); // TODO: Residu

	// Topology
	char* topology = argv[6];
	if (strcmp(topology, "star") == 0) {
		SELECTED_TOPOLOGY = STAR;
		printf("\nTopology: %s", "star");
	} else if (strcmp(topology, "chain") == 0) {
		SELECTED_TOPOLOGY = CHAIN;
		printf("\nTopology: %s", "chain");
	} else if (strcmp(topology, "circle") == 0) {
		SELECTED_TOPOLOGY = CIRCLE;
		printf("\nTopology: %s", "circle");
	} else if (strcmp(topology, "mesh") == 0) {
		SELECTED_TOPOLOGY = MESH;
		printf("\nTopology: %s", "mesh");
	} else {
		printf("\nGiven topology type is undefined.");
		exit(1);
	}

	// Bridge
	char* bridge = argv[7];
	if (strcmp(bridge, "high_degree") == 0) {
		SELECTED_BRIDGE_NODE_SELECTION = HIGH_DEGREE_NODES;
		printf("\nBridge: %s", "high degree");
	} else if (strcmp(bridge, "random") == 0) {
		SELECTED_BRIDGE_NODE_SELECTION = RANDOM_NODES;
		printf("\nBridge: %s", "random");
	} else {
		printf("\nGiven bridge type is undefined.");
		exit(1);
	}

	//  Interconnection
	sscanf(argv[8], "%d", &AMOUNT_INTERCONNECTIONS);
	printf("\nAmount of interconnection: %d", AMOUNT_INTERCONNECTIONS);

	// Force undirected (TODO: Should be optional)
	char* force_undirected = argv[9];
	if (strcmp(force_undirected, "undirected") == 0) {
		FORCE_UNDIRECTED_BRIDGES = true;
		printf("\nUndirected bridges added.");
	}
}

void sample_graph(char* input_path, char* output_path, float fraction) {
	std::vector<int> source_vertices;
	std::vector<int> destination_vertices;

	// Convert edge list to COO
	COO_List* coo_list = load_graph_from_edge_list_file_to_coo(source_vertices, destination_vertices, input_path);
	
	// Convert the COO graph into a CSR format (for the in-memory GPU representation) 
	CSR_List* csr_list = convert_coo_to_csr_format(coo_list->source, coo_list->destination);

	// Edge based Node Sampling Step
	Sampled_Vertices* sampled_vertices = perform_edge_based_node_sampling_step(coo_list->source, coo_list->destination, fraction);
	printf("\nCollected %d vertices.", sampled_vertices->sampled_vertices_size);

	// Induction step (TODO: re-use device memory from CSR conversion)
	int* d_offsets;
	int* d_indices;
	gpuErrchk(hipMalloc((void**)&d_offsets, sizeof(int)*(SIZE_VERTICES + 1)));
	gpuErrchk(hipMalloc((void**)&d_indices, sizeof(int)*SIZE_EDGES));
	gpuErrchk(hipMemcpy(d_indices, csr_list->indices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_offsets, csr_list->offsets, sizeof(int)*(SIZE_VERTICES + 1), hipMemcpyHostToDevice));

	int* d_sampled_vertices;
	gpuErrchk(hipMalloc((void**)&d_sampled_vertices, sizeof(int)*SIZE_VERTICES));
	gpuErrchk(hipMemcpy(d_sampled_vertices, sampled_vertices->vertices, sizeof(int)*(SIZE_VERTICES), hipMemcpyHostToDevice));

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(D_SIZE_EDGES), &SIZE_EDGES, sizeof(int), 0, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(D_SIZE_VERTICES), &SIZE_VERTICES, sizeof(int), 0, hipMemcpyHostToDevice));

	Edge* d_edge_data;
	gpuErrchk(hipMalloc((void**)&d_edge_data, sizeof(Edge)*SIZE_EDGES));

	printf("\nRunning kernel (induction step) with block size %d and thread size %d:", get_block_size(), get_thread_size());
	perform_induction_step <<<get_block_size(), get_thread_size() >> >(d_sampled_vertices, d_offsets, d_indices, d_edge_data);

	int h_edge_count;
	gpuErrchk(hipMemcpyFromSymbol(&h_edge_count, HIP_SYMBOL(d_edge_count), sizeof(int)));
	if (h_edge_count >= SIZE_EDGES + 1) {
		printf("overflow error\n"); return;
	}

	printf("\nAmount of edges collected: %d", h_edge_count);
	std::vector<Edge> results(h_edge_count);
	gpuErrchk(hipMemcpy(&(results[0]), d_edge_data, h_edge_count * sizeof(Edge), hipMemcpyDeviceToHost));
	
	write_output_to_file(results, output_path);

	hipFree(d_offsets);
	hipFree(d_indices);
	hipFree(d_sampled_vertices);
	
	// Cleanup
	free(sampled_vertices->vertices);
	free(sampled_vertices);

	free(coo_list);

	free(csr_list->indices);
	free(csr_list->offsets);
	free(csr_list);
}

/*
Fast conversion to CSR - Using nvGraph for conversion
Modified from: github.com/bmass02/nvGraphExample
*/
CSR_List* convert_coo_to_csr_format(int* source_vertices, int* target_vertices) {
	printf("\nConverting COO to CSR format.");
	CSR_List* csr_list = (CSR_List*)malloc(sizeof(CSR_List));
	csr_list->offsets = (int*)malloc((SIZE_VERTICES + 1) * sizeof(int));
	csr_list->indices = (int*)malloc(SIZE_EDGES * sizeof(int));

	// First setup the COO format from the input (source_vertices and target_vertices array)
	nvgraphHandle_t handle;
	nvgraphGraphDescr_t graph;
	nvgraphCreate(&handle);
	nvgraphCreateGraphDescr(handle, &graph);
	nvgraphCOOTopology32I_t cooTopology = (nvgraphCOOTopology32I_t)malloc(sizeof(struct nvgraphCOOTopology32I_st));
	cooTopology->nedges = SIZE_EDGES;
	cooTopology->nvertices = SIZE_VERTICES;
	cooTopology->tag = NVGRAPH_UNSORTED;

	gpuErrchk(hipMalloc((void**)&cooTopology->source_indices, SIZE_EDGES * sizeof(int)));
	gpuErrchk(hipMalloc((void**)&cooTopology->destination_indices, SIZE_EDGES * sizeof(int)));

	gpuErrchk(hipMemcpy(cooTopology->source_indices, source_vertices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(cooTopology->destination_indices, target_vertices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice));

	// Edge data (allocated, but not used)
	hipDataType data_type = HIP_R_32F;
	float* d_edge_data;
	float* d_destination_edge_data;
	gpuErrchk(hipMalloc((void**)&d_edge_data, sizeof(float) * SIZE_EDGES)); // Note: only allocate this for 1 float since we don't have any data yet
	gpuErrchk(hipMalloc((void**)&d_destination_edge_data, sizeof(float) * SIZE_EDGES)); // Note: only allocate this for 1 float since we don't have any data yet

	nvgraphCSRTopology32I_t csrTopology = (nvgraphCSRTopology32I_t)malloc(sizeof(struct nvgraphCSRTopology32I_st));
	int **d_indices = &(csrTopology->destination_indices);
	int **d_offsets = &(csrTopology->source_offsets);

	gpuErrchk(hipMalloc((void**)d_indices, SIZE_EDGES * sizeof(int)));
	gpuErrchk(hipMalloc((void**)d_offsets, (SIZE_VERTICES + 1) * sizeof(int)));

	check(nvgraphConvertTopology(handle, NVGRAPH_COO_32, cooTopology, d_edge_data, &data_type, NVGRAPH_CSR_32, csrTopology, d_destination_edge_data));

	gpuErrchk(hipPeekAtLastError());

	// Copy data to the host (without edge data)
	gpuErrchk(hipMemcpy(csr_list->indices, *d_indices, SIZE_EDGES * sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(csr_list->offsets, *d_offsets, (SIZE_VERTICES + 1) * sizeof(int), hipMemcpyDeviceToHost));

	// Clean up (Data allocated on device and both topologies, since we only want to work with indices and offsets for now)
	hipFree(d_indices);
	hipFree(d_offsets);
	hipFree(d_edge_data);
	hipFree(d_destination_edge_data);
	hipFree(cooTopology->destination_indices);
	hipFree(cooTopology->source_indices);
	free(cooTopology);
	free(csrTopology);

	return csr_list;
}

int get_thread_size() {
	return ((SIZE_VERTICES + 1) > MAX_THREADS) ? MAX_THREADS : SIZE_VERTICES;
}

int get_block_size() {
	return ((SIZE_VERTICES + 1) > MAX_THREADS) ? ((SIZE_VERTICES / MAX_THREADS) + 1) : 1;
}

int calculate_node_sampled_size(float fraction) {
	return int(SIZE_VERTICES * fraction);
}

/*
NOTE: Only reads integer vertices for now (through the 'sscanf' function) and obvious input vertices arrays
*/
void load_graph_from_edge_list_file(int* source_vertices, int* target_vertices, char* file_path) {
	printf("\nLoading graph file from: %s", file_path);

	FILE* file = fopen(file_path, "r");
	char line[256];
	int edge_index = 0;

	while (fgets(line, sizeof(line), file)) {
		if (line[0] == '#') {
			//print_debug_log("\nEscaped a comment.");
			continue;
		}

		// Save source and target vertex (temp)
		int source_vertex;
		int target_vertex;

		sscanf(line, "%d%d\t", &source_vertex, &target_vertex);

		// Add vertices to the source and target arrays, forming an edge accordingly
		source_vertices[edge_index] = source_vertex;
		target_vertices[edge_index] = target_vertex;

		// Increment edge index to add any new edge
		edge_index++;

		//print_debug_log("\nAdded start vertex:", source_vertex);
		//print_debug_log("\nAdded end vertex:", target_vertex);
	}

	fclose(file);
}

COO_List* load_graph_from_edge_list_file_to_coo(std::vector<int>& source_vertices, std::vector<int>& destination_vertices, char* file_path) {
	printf("\nLoading graph file from: %s", file_path);

	FILE* file = fopen(file_path, "r");

	char line[256];

	int current_coordinate = 0;
	if (IS_INPUT_FILE_COO) { // Saves many 'if' ticks inside the while loop - If the input file is already a COO, simply add the coordinates the vectors.
		std::unordered_set<int> vertices;
		
		while (fgets(line, sizeof(line), file)) {
			if (line[0] == '#' || line[0] == '\n') {
				//print_debug_log("\nEscaped a comment.");
				continue;
			}

			// Save source and target vertex (temp)
			int source_vertex;
			int target_vertex;

			sscanf(line, "%d%d\t", &source_vertex, &target_vertex);

			// Add vertices to the source and target arrays, forming an edge accordingly
			source_vertices.push_back(source_vertex);
			destination_vertices.push_back(target_vertex);
			vertices.insert(source_vertex);
			vertices.insert(target_vertex);
		}

		SIZE_VERTICES = vertices.size();
		SIZE_EDGES = source_vertices.size();

		printf("\nTotal amount of vertices: %zd", SIZE_VERTICES);
		printf("\nTotal amount of edges: %zd", SIZE_EDGES);
	} else {
		std::unordered_map<int, int> map_from_edge_to_coordinate;

		while (fgets(line, sizeof(line), file)) {
			if (line[0] == '#' || line[0] == '\n') {
				//print_debug_log("\nEscaped a comment.");
				continue;
			}

			// Save source and target vertex (temp)
			int source_vertex;
			int target_vertex;

			sscanf(line, "%d%d\t", &source_vertex, &target_vertex);

			// Add vertices to the source and target arrays, forming an edge accordingly
			current_coordinate = add_vertex_as_coordinate(source_vertices, map_from_edge_to_coordinate, source_vertex, current_coordinate);
			current_coordinate = add_vertex_as_coordinate(destination_vertices, map_from_edge_to_coordinate, target_vertex, current_coordinate);
		}

		SIZE_VERTICES = map_from_edge_to_coordinate.size();
		SIZE_EDGES = source_vertices.size();

		printf("\nTotal amount of vertices: %zd", SIZE_VERTICES);
		printf("\nTotal amount of edges: %zd", SIZE_EDGES);
	}

	COO_List* coo_list = (COO_List*)malloc(sizeof(COO_List));

	source_vertices.reserve(source_vertices.size());
	destination_vertices.reserve(destination_vertices.size());
	coo_list->source = &source_vertices[0];
	coo_list->destination = &destination_vertices[0];

	if (source_vertices.size() != destination_vertices.size()) {
		printf("\nThe size of the source vertices does not equal the destination vertices.");
		exit(1);
	}

	bool SAVE_INPUT_FILE_AS_COO = false;
	if (SAVE_INPUT_FILE_AS_COO) {
		save_input_file_as_coo(source_vertices, destination_vertices, "C:\\Users\\AJ\\Desktop\\new_datasets\\coo\\none.txt");
	}

	// Print edges
	/*for (int i = 0; i < source_vertices.size(); i++) {
	printf("\n(%d, %d)", coo_list->source[i], coo_list->destination[i]);
	}*/

	fclose(file);

	return coo_list;
}

int add_vertex_as_coordinate(std::vector<int>& vertices_type, std::unordered_map<int, int>& map_from_edge_to_coordinate, int vertex, int coordinate) {
	if (map_from_edge_to_coordinate.count(vertex)) {
		vertices_type.push_back(map_from_edge_to_coordinate.at(vertex));

		return coordinate;
	}
	else {
		map_from_edge_to_coordinate[vertex] = coordinate;
		vertices_type.push_back(coordinate);
		coordinate++;

		return coordinate;
	}
}

Sampled_Vertices* perform_edge_based_node_sampling_step(int* source_vertices, int* target_vertices, float fraction) {
	printf("\nPerforming edge based node sampling step.\n");

	Sampled_Vertices* sampled_vertices = (Sampled_Vertices*)malloc(sizeof(Sampled_Vertices));

	int amount_total_sampled_vertices = calculate_node_sampled_size(fraction);

	std::random_device seeder;
	std::mt19937 engine(seeder());

	sampled_vertices->vertices = (int*)calloc(SIZE_VERTICES, sizeof(int));
	int collected_amount = 0;

	while (collected_amount < amount_total_sampled_vertices) {
		// Pick a random vertex u
		std::uniform_int_distribution<int> range_edges(0, (SIZE_EDGES - 1)); // Don't select the last element in the offset
		int random_edge_index = range_edges(engine);
		
		// Insert u, v (TODO: extract to method per vertex)
		if (!sampled_vertices->vertices[source_vertices[random_edge_index]]) {
			sampled_vertices->vertices[source_vertices[random_edge_index]] = 1;
			print_debug_log("\nCollected vertex:", source_vertices[random_edge_index]);
			//printf("\nCollected vertex: %d", source_vertices[random_edge_index]);
			collected_amount++;
		}
		if (!sampled_vertices->vertices[target_vertices[random_edge_index]]) {
			sampled_vertices->vertices[target_vertices[random_edge_index]] = 1;
			print_debug_log("\nCollected vertex:", target_vertices[random_edge_index]);
			//printf("\nCollected vertex: %d", target_vertices[random_edge_index]);
			collected_amount++;
		}
	}

	sampled_vertices->sampled_vertices_size = collected_amount;

	printf("\nDone with node sampling step..");

	return sampled_vertices;
}

/*
=======================================================================================
Expanding code
=======================================================================================
*/

void expand_graph(char* input_path, char* output_path, float scaling_factor) {
	std::vector<int> source_vertices;
	std::vector<int> destination_vertices;
	COO_List* coo_list = load_graph_from_edge_list_file_to_coo(source_vertices, destination_vertices, input_path);
	CSR_List* csr_list = convert_coo_to_csr_format(coo_list->source, coo_list->destination);

	const int amount_of_sampled_graphs = scaling_factor / SAMPLING_FRACTION;
	printf("Amount of sampled graph versions: %d", amount_of_sampled_graphs);

	Sampled_Vertices** sampled_vertices_per_graph = (Sampled_Vertices**)malloc(sizeof(Sampled_Vertices)*amount_of_sampled_graphs);

	int** d_size_collected_edges = (int**)malloc(sizeof(int*)*amount_of_sampled_graphs);
	Edge** d_edge_data_expanding = (Edge**)malloc(sizeof(Edge*)*amount_of_sampled_graphs);

	Sampled_Graph_Version* sampled_graph_version_list = new Sampled_Graph_Version[amount_of_sampled_graphs];
	char current_label = 'a';

	// Sequential version
	for (int i = 0; i < amount_of_sampled_graphs; i++) {
		sampled_vertices_per_graph[i] = perform_edge_based_node_sampling_step(coo_list->source, coo_list->destination, SAMPLING_FRACTION);
		printf("\nCollected %d vertices.", sampled_vertices_per_graph[i]->sampled_vertices_size);
		
		std::vector<Edge> edges;
		perform_sequential_induction_step(sampled_vertices_per_graph[i]->vertices, csr_list->offsets, csr_list->indices, edges);

		Sampled_Graph_Version* sampled_graph_version = new Sampled_Graph_Version();
		(*sampled_graph_version).edges = edges; // Mweh

		// Label
		sampled_graph_version->label = current_label++;

		// Copy data to the sampled version list
		sampled_graph_version_list[i] = (*sampled_graph_version);

		// Cleanup
		delete(sampled_graph_version);
		free(sampled_vertices_per_graph[i]->vertices);
		free(sampled_vertices_per_graph[i]);
	}
	
	// Parallell version (GPU CODE)
	/*
	int* d_offsets;
	int* d_indices;
	gpuErrchk(hipMalloc((void**)&d_offsets, sizeof(int)*(SIZE_VERTICES + 1)));
	gpuErrchk(hipMalloc((void**)&d_indices, sizeof(int)*SIZE_EDGES));

	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(D_SIZE_EDGES), &SIZE_EDGES, sizeof(int), 0, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(D_SIZE_VERTICES), &SIZE_VERTICES, sizeof(int), 0, hipMemcpyHostToDevice));

	for (int i = 0; i < amount_of_sampled_graphs; i++) {
		sampled_vertices_per_graph[i] = perform_edge_based_node_sampling_step(coo_list->source, coo_list->destination, SAMPLING_FRACTION);
		printf("\nCollected %d vertices.", sampled_vertices_per_graph[i]->sampled_vertices_size);

		gpuErrchk(hipMemcpy(d_indices, csr_list->indices, SIZE_EDGES * sizeof(int), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_offsets, csr_list->offsets, sizeof(int)*(SIZE_VERTICES + 1), hipMemcpyHostToDevice));

		int* d_sampled_vertices;
		gpuErrchk(hipMalloc((void**)&d_sampled_vertices, sizeof(int)*SIZE_VERTICES));
		gpuErrchk(hipMemcpy(d_sampled_vertices, sampled_vertices_per_graph[i]->vertices, sizeof(int)*(SIZE_VERTICES), hipMemcpyHostToDevice));

		int* h_size_edges = 0;
		gpuErrchk(hipMalloc((void**)&d_size_collected_edges[i], sizeof(int)));
		gpuErrchk(hipMemcpy(d_size_collected_edges[i], &h_size_edges, sizeof(int), hipMemcpyHostToDevice));

		gpuErrchk(hipMalloc((void**)&d_edge_data_expanding[i], sizeof(Edge)*SIZE_EDGES));

		hipDeviceSynchronize(); // This can be deleted - double check

		printf("\nRunning kernel (induction step) with block size %d and thread size %d:", get_block_size(), get_thread_size());
		perform_induction_step_expanding<<<get_block_size(), get_thread_size() >> >(d_sampled_vertices, d_offsets, d_indices, d_edge_data_expanding[i], d_size_collected_edges[i]);

		// Edge size
		int h_size_edges_result;
		gpuErrchk(hipMemcpy(&h_size_edges_result, d_size_collected_edges[i], sizeof(int), hipMemcpyDeviceToHost));

		// Edges
		printf("\nh_size_edges: %d", h_size_edges_result);
		Sampled_Graph_Version* sampled_graph_version = new Sampled_Graph_Version();
		(*sampled_graph_version).edges.resize(h_size_edges_result);

		gpuErrchk(hipMemcpy(&sampled_graph_version->edges[0], d_edge_data_expanding[i], sizeof(Edge)*(h_size_edges_result), hipMemcpyDeviceToHost));

		// Label
		sampled_graph_version->label = current_label++;

		// Copy data to the sampled version list
		sampled_graph_version_list[i] = (*sampled_graph_version);

		// Cleanup
		delete(sampled_graph_version);

		hipFree(d_sampled_vertices);
		hipFree(d_edge_data_expanding[i]);
		hipFree(d_size_collected_edges);
		free(sampled_vertices_per_graph[i]->vertices);
		free(sampled_vertices_per_graph[i]);
	}

	hipFree(d_offsets);
	hipFree(d_indices);
	free(sampled_vertices_per_graph);
	free(coo_list);
	free(csr_list->indices);
	free(csr_list->offsets);
	free(csr_list);*/
	
	// For each sampled graph version, copy the data back to the host
	std::vector<Bridge_Edge> bridge_edges;

	switch (SELECTED_TOPOLOGY) {
		case STAR:
			link_using_star_topology(sampled_graph_version_list, amount_of_sampled_graphs, bridge_edges);
			break;
		case CHAIN:
			link_using_line_topology(sampled_graph_version_list, amount_of_sampled_graphs, bridge_edges);
			break;
		case CIRCLE:
			link_using_circle_topology(sampled_graph_version_list, amount_of_sampled_graphs, bridge_edges);
			break;
		case MESH:
			link_using_mesh_topology(sampled_graph_version_list, amount_of_sampled_graphs, bridge_edges);
	}

	printf("\nConnected by adding a total of %d bridge edges.", bridge_edges.size());

	write_expanded_output_to_file(sampled_graph_version_list, amount_of_sampled_graphs, bridge_edges, output_path);

	// Cleanup
	delete[] sampled_graph_version_list;
}

void link_using_star_topology(Sampled_Graph_Version* sampled_graph_version_list, int amount_of_sampled_graphs, std::vector<Bridge_Edge>& bridge_edges) {
	Sampled_Graph_Version center_graph = sampled_graph_version_list[0]; // First sampled version will be the graph in the center

	for (int i = 1; i < amount_of_sampled_graphs; i++) { // Skip the center graph 
		add_edge_interconnection_between_graphs(&(sampled_graph_version_list[i]), &center_graph, bridge_edges);
	}

}

void link_using_line_topology(Sampled_Graph_Version* sampled_graph_version_list, int amount_of_sampled_graphs, std::vector<Bridge_Edge>& bridge_edges) {
	for (int i = 0; i < (amount_of_sampled_graphs-1); i++) {
		add_edge_interconnection_between_graphs(&(sampled_graph_version_list[i]), &(sampled_graph_version_list[i+1]), bridge_edges);
	}
}

void link_using_circle_topology(Sampled_Graph_Version* sampled_graph_version_list, int amount_of_sampled_graphs, std::vector<Bridge_Edge>& bridge_edges) {
	for (int i = 0; i < amount_of_sampled_graphs; i++) {
		if (i == (amount_of_sampled_graphs-1)) { // We're at the last sampled graph, so connect it back to the first one in the list
			add_edge_interconnection_between_graphs(&(sampled_graph_version_list[i]), &(sampled_graph_version_list[0]), bridge_edges);
			break;
		}

		add_edge_interconnection_between_graphs(&(sampled_graph_version_list[i]), &(sampled_graph_version_list[i+1]), bridge_edges);
	}
}

void link_using_mesh_topology(Sampled_Graph_Version* sampled_graph_version_list, int amount_of_sampled_graphs, std::vector<Bridge_Edge>& bridge_edges) {
	for (int x = 0; x < amount_of_sampled_graphs; x++) {
		Sampled_Graph_Version current_graph = sampled_graph_version_list[x];

		for (int y = 0; y < amount_of_sampled_graphs; y++) {
			if (x==y) { // Don't link the current graph to itself
				continue;
			}

			add_edge_interconnection_between_graphs(&(sampled_graph_version_list[x]), &(sampled_graph_version_list[y]), bridge_edges);
		}
	}
}

/*
-> Probably parallelizable.
-> if(amount_of_edge_interconnections<1) = fraction of the edges/nodes?
*/
void add_edge_interconnection_between_graphs(Sampled_Graph_Version* graph_a, Sampled_Graph_Version* graph_b, std::vector<Bridge_Edge>& bridge_edges) {
	for (int i = 0; i < AMOUNT_INTERCONNECTIONS; i++) {
		int vertex_a = get_node_bridge_vertex(graph_a);
		int vertex_b = get_node_bridge_vertex(graph_b);

		// Add edge
		Bridge_Edge bridge_edge;
		sprintf(bridge_edge.source, "%c%d", graph_a->label, vertex_a);
		sprintf(bridge_edge.destination, "%c%d", graph_b->label, vertex_b);
		bridge_edges.push_back(bridge_edge);
		//printf("\nBridge selection - Selected: (%s, %s)", bridge_edge.source, bridge_edge.destination);

		if (FORCE_UNDIRECTED_BRIDGES) {
			Bridge_Edge bridge_edge_undirected;
			sprintf(bridge_edge_undirected.source, "%c%d", graph_b->label, vertex_b);
			sprintf(bridge_edge_undirected.destination, "%c%d", graph_a->label, vertex_a);
			bridge_edges.push_back(bridge_edge_undirected);
			//printf("\nBridge selection (undirected) - Selected: (%s, %s)", bridge_edge_undirected.source, bridge_edge_undirected.destination);
		}
	}
}

// TODO: Add parameter (e.g. Random/high-degree nodes/low-degree nodes)
int select_random_bridge_vertex(Sampled_Graph_Version* graph) {
	// TODO: Move to add_edge_interconnection_between_graphs
	std::random_device seeder;
	std::mt19937 engine(seeder());
	std::uniform_int_distribution<int> range_edges(0, ((*graph).edges.size()) - 1);
	int random_edge_index = range_edges(engine);

	// 50:50 return source or destination
	std::random_device destination_or_source_seeder;
	std::mt19937 engine_source_or_destination(destination_or_source_seeder());
	std::uniform_int_distribution<int> range_destination_source(0, 1);
	int destination_or_source = range_destination_source(engine_source_or_destination);

	if (destination_or_source == 0) {
		return (*graph).edges[random_edge_index].source; 
	}
	else {
		return (*graph).edges[random_edge_index].destination; 
	}
}

int select_high_degree_node_bridge_vertex(Sampled_Graph_Version* graph) {
	if (graph->high_degree_nodes.size() > 0) { // There already exists some high degree nodes here, so just select them randomly for instance.
		return get_random_high_degree_node(graph);
	} else { // Collect high degree nodes and add them to the current graph
		// Map all vertices onto a map along with their degree
		std::unordered_map<int, int> node_degree;
		
		for (auto &edge : graph->edges) {
			++node_degree[edge.source];
			++node_degree[edge.destination];
		}

		// Convert the map to a vector
		std::vector<std::pair<int, int>> node_degree_vect(node_degree.begin(), node_degree.end());

		// Sort the vector (ascending, high degree nodes are on top)
		std::sort(node_degree_vect.begin(), node_degree_vect.end(), [](const std::pair<int, int> &left, const std::pair<int, int> &right) {
			return left.second > right.second;
		});

		// Collect only the nodes (half of the total nodes) that have a high degree
		for (int i = 0; i < node_degree_vect.size() / 2; i++) {
			graph->high_degree_nodes.push_back(node_degree_vect[i].first);
		}

		return get_random_high_degree_node(graph);
	}
}

int get_random_high_degree_node(Sampled_Graph_Version* graph) {
	std::random_device seeder;
	std::mt19937 engine(seeder());

	std::uniform_int_distribution<int> range_edges(0, (graph->high_degree_nodes.size() - 1));
	int random_vertex_index = range_edges(engine);

	return graph->high_degree_nodes[random_vertex_index];
}

int get_node_bridge_vertex(Sampled_Graph_Version* graph) {
	switch (SELECTED_BRIDGE_NODE_SELECTION) {
		case RANDOM_NODES:
			return select_random_bridge_vertex(graph);
		case HIGH_DEGREE_NODES:
			return select_high_degree_node_bridge_vertex(graph);
	}
}

void write_expanded_output_to_file(Sampled_Graph_Version* sampled_graph_version_list, int amount_of_sampled_graphs, std::vector<Bridge_Edge>& bridge_edges, char* ouput_path) {
	printf("\nWriting results to output file.");

	char* file_path = ouput_path;
	FILE *output_file = fopen(file_path, "w");

	if (output_file == NULL) {
		printf("\nError writing results to output file.");
		exit(1);
	}

	// Write sampled graph versions
	for (int i = 0; i < amount_of_sampled_graphs; i++) {
		for (int p = 0; p < sampled_graph_version_list[i].edges.size(); p++) {
			fprintf(output_file, "\n%c%d\t%c%d", sampled_graph_version_list[i].label, sampled_graph_version_list[i].edges[p].source, sampled_graph_version_list[i].label, sampled_graph_version_list[i].edges[p].destination);
		}
	}

	for (int i = 0; i < bridge_edges.size(); i++) {
		fprintf(output_file, "\n%s\t%s", bridge_edges[i].source, bridge_edges[i].destination);
	}

	fclose(output_file);
}

void write_output_to_file(std::vector<Edge>& results, char* output_path) {
	printf("\nWriting results to output file.");
	
	char* file_path = output_path;
	FILE *output_file = fopen(file_path, "w");

	if (output_file == NULL) {
		printf("\nError writing results to output file.");
		exit(1);
	}

	for (int i = 0; i < results.size(); i++) {
		fprintf(output_file, "%d\t%d\n", results[i].source, results[i].destination);
	}

	fclose(output_file);
}


void save_input_file_as_coo(std::vector<int>& source_vertices, std::vector<int>& destination_vertices, char* save_path) {
	printf("\nWriting results to output file.");

	char* file_path = save_path;
	FILE *output_file = fopen(file_path, "w");

	if (output_file == NULL) {
		printf("\nError writing results to output file.");
		exit(1);
	}

	for (int i = 0; i < source_vertices.size(); i++) {
		fprintf(output_file, "%d\t%d\n", source_vertices[i], destination_vertices[i]);
	}

	fclose(output_file);
}

void print_coo(int* source_vertices, int* end_vertices) {
	for (int i = 0; i < SIZE_EDGES; i++) {
		printf("\n%d, %d", source_vertices[i], end_vertices[i]);
	}
}

void print_csr(int* h_offsets, int* h_indices) {
	printf("\nRow Offsets (Vertex Table):\n");
	for (int i = 0; i < SIZE_VERTICES + 1; i++) {
		printf("%d, ", h_offsets[i]);
	}

	printf("\nColumn Indices (Edge Table):\n");
	for (int i = 0; i < SIZE_EDGES; i++) {
		printf("%d, ", h_indices[i]);
	}
}

void check(nvgraphStatus_t status) {
	if (status == NVGRAPH_STATUS_NOT_INITIALIZED) {
		printf("\nError converting to CSR: %d - NVGRAPH_STATUS_NOT_INITIALIZED", status);
		exit(0);
	}
	else if (status == NVGRAPH_STATUS_ALLOC_FAILED) {
		printf("\nError converting to CSR: %d - NVGRAPH_STATUS_ALLOC_FAILED", status);
		exit(0);
	}
	else if (status == NVGRAPH_STATUS_INVALID_VALUE) {
		printf("\nError converting to CSR: %d - NVGRAPH_STATUS_INVALID_VALUE", status);
		exit(0);
	}
	else if (status == NVGRAPH_STATUS_INTERNAL_ERROR) {
		printf("\nError converting to CSR: %d - NVGRAPH_STATUS_INTERNAL_ERROR", status);
		exit(0);
	}
	else if (status == NVGRAPH_STATUS_SUCCESS) {
		printf("\nConverted to CSR successfully (statuscode %d).\n", status);
	}
	else {
		printf("\nSome other error occurred while trying to convert to CSR.");
		exit(0);
	}
}

void print_debug_log(char* message) {
	if (ENABLE_DEBUG_LOG)
		printf("%s", message);
}

void print_debug_log(char* message, int value) {
	if (ENABLE_DEBUG_LOG)
		printf("%s %d", message, value);
}